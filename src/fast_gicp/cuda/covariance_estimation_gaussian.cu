#include "hip/hip_runtime.h"
#include <fast_gicp/cuda/covariance_estimation.cuh>

#include <thrust/device_vector.h>

#include <thrust/async/for_each.h>
#include <thrust/async/transform.h>

namespace fast_gicp {
namespace cuda {

struct NormalDistribution {
public:
  EIGEN_MAKE_ALIGNED_OPERATOR_NEW

  __host__ __device__ NormalDistribution() {}

  static __host__ __device__ NormalDistribution zero() {
    NormalDistribution dist;
    dist.sum_weights = 0.0f;
    dist.mean.setZero();
    dist.cov.setZero();
    return dist;
  }

  __host__ __device__ NormalDistribution operator+(const NormalDistribution& rhs) const {
    NormalDistribution sum;
    sum.sum_weights = sum_weights + rhs.sum_weights;
    sum.mean = mean + rhs.mean;
    sum.cov = cov + rhs.cov;
    return sum;
  }

  __host__ __device__ NormalDistribution& operator+=(const NormalDistribution& rhs) {
    sum_weights += rhs.sum_weights;
    mean += rhs.mean;
    cov += rhs.cov;
    return *this;
  }

  __host__ __device__ void accumulate(const float w, const Eigen::Vector3f& x) {
    sum_weights += w;
    mean += w * x;
    cov += w * x * x.transpose();
  }

  __host__ __device__ NormalDistribution& finalize() {
    Eigen::Vector3f sum_pt = mean;
    mean /= sum_weights;
    cov = (cov - mean * sum_pt.transpose()) / sum_weights;

    return *this;
  }

  float sum_weights;
  Eigen::Vector3f mean;
  Eigen::Matrix3f cov;
};


struct covariance_estimation_kernel_gaussian {
  static const int BLOCK_SIZE = 512;

  covariance_estimation_kernel_gaussian(thrust::device_ptr<const float> exp_factor_ptr, thrust::device_ptr<const float> max_dist_ptr, thrust::device_ptr<const Eigen::Vector3f> points_ptr)
  : exp_factor_ptr(exp_factor_ptr),
    max_dist_ptr(max_dist_ptr),
    points_ptr(points_ptr) {}

  __host__ __device__ NormalDistribution operator()(const Eigen::Vector3f& x) const {
    const float exp_factor = *thrust::raw_pointer_cast(exp_factor_ptr);
    const float max_dist = *thrust::raw_pointer_cast(max_dist_ptr);
    const float max_dist_sq = max_dist * max_dist;
    const Eigen::Vector3f* points = thrust::raw_pointer_cast(points_ptr);

    NormalDistribution dist = NormalDistribution::zero();
    for (int i = 0; i < BLOCK_SIZE; i++) {
      float sq_d = (x - points[i]).squaredNorm();
      if (sq_d > max_dist_sq) {
        continue;
      }

      float r = sqrt(sq_d);
      float w = expf(-r * r / (2 * exp_factor * exp_factor));
      dist.accumulate(w, points[i]);
    }

    return dist;
  }

  thrust::device_ptr<const float> exp_factor_ptr;
  thrust::device_ptr<const float> max_dist_ptr;
  thrust::device_ptr<const Eigen::Vector3f> points_ptr;
};

struct finalization_kernel {
  finalization_kernel(const int stride, const thrust::device_vector<NormalDistribution>& accumulated_dists)
  : stride(stride),
    accumulated_dists_first(accumulated_dists.data()),
    accumulated_dists_last(accumulated_dists.data() + accumulated_dists.size()) {}

  __host__ __device__ Eigen::Matrix3f operator()(int index) const {
    const NormalDistribution* dists = thrust::raw_pointer_cast(accumulated_dists_first);
    const NormalDistribution* dists_last = thrust::raw_pointer_cast(accumulated_dists_last);
    const int num_dists = dists_last - dists;

    NormalDistribution sum = dists[index];
    for (int dist_index = index + stride; dist_index < num_dists; dist_index += stride) {
      sum += dists[dist_index];
    }

    return sum.finalize().cov;
  }

  const int stride;
  thrust::device_ptr<const NormalDistribution> accumulated_dists_first;
  thrust::device_ptr<const NormalDistribution> accumulated_dists_last;
};

void covariance_estimation_gaussian(const thrust::device_vector<Eigen::Vector3f>& points, double kernel_width, double max_dist, thrust::device_vector<Eigen::Matrix3f>& covariances) {
  covariances.resize(points.size());

  thrust::device_vector<float> constants(2);
  constants[0] = kernel_width;
  constants[1] = max_dist;
  thrust::device_ptr<const float> exp_factor_ptr = constants.data();
  thrust::device_ptr<const float> max_dist_ptr = constants.data() + 1;

  int num_blocks = (points.size() + (covariance_estimation_kernel_gaussian::BLOCK_SIZE - 1)) / covariance_estimation_kernel_gaussian::BLOCK_SIZE;
  // padding
  thrust::device_vector<Eigen::Vector3f> ext_points(num_blocks * covariance_estimation_kernel_gaussian::BLOCK_SIZE);
  thrust::copy(points.begin(), points.end(), ext_points.begin());
  thrust::fill(ext_points.begin() + points.size(), ext_points.end(), Eigen::Vector3f(0.0f, 0.0f, 0.0f));

  thrust::device_vector<NormalDistribution> accumulated_dists(points.size() * num_blocks);

  thrust::system::cuda::detail::unique_stream stream;
  std::vector<thrust::system::cuda::unique_eager_event> events(num_blocks);

  // accumulate kerneled point distributions
  for (int i = 0; i < num_blocks; i++) {
    covariance_estimation_kernel_gaussian kernel(exp_factor_ptr, max_dist_ptr, ext_points.data() + covariance_estimation_kernel_gaussian::BLOCK_SIZE * i);
    auto event = thrust::async::transform(points.begin(), points.end(), accumulated_dists.begin() + points.size() * i, kernel);
    events[i] = std::move(event);
    thrust::system::cuda::detail::create_dependency(stream, events[i]);
  }

  // finalize distributions
  thrust::transform(
    thrust::cuda::par.on(stream.native_handle()),
    thrust::counting_iterator<int>(0),
    thrust::counting_iterator<int>(points.size()),
    covariances.begin(),
    finalization_kernel(points.size(), accumulated_dists));
}

}  // namespace cuda
}  // namespace fast_gicp